
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>

__global__ void manipulateSystem(bool *system, int M, int N) {
    int i, j, x, y, index, testCase = 0, neighbors = 0;
    bool temp;
    index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index == 0) {                           //top left
        testCase = 5;
    } else if (index == M - 1){                 //top right
        testCase = 6;
    } else if (index == (M - 1) * N) {          //bottom left
        testCase = 7;
    } else if (index == M * N - 1) {            //bottom right
        testCase = 8;
    } else if (index % M == 0) {                //left
        testCase = 1;
    } else if (index % M == M - 1) {            //right
        testCase = 2;
    } else if (index > 0 && index < M) {        //top
        testCase = 3;
    } else if (index > (M - 1) * N) {           //bottom
        testCase = 4;
    } else {
        testCase = 0;
    }

    __syncthreads();
    switch (testCase) {
        case 0:         //center 
            for (i = index - M - 1; i < index + M; i+=M) {
                for (j = 0; j < 3; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
        case 1:         //left
            for (i = index - M - 1; i < index + M; i += M) {
                for (j = 1; j < 3; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
        case 2:         //right
            for (i = index - M - 1; i < index + M; i += M) {
                for (j = 0; j < 2; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
        case 3:         //top
            for (i = index - 1; i < index + M; i += M) {
                for (j = 0; j < 3; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
        case 4:         //bottom
            for (i = index - M - 1; i < index; i += M) {
                for (j = 0; j < 3; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
        case 5:         //top left
            for (i = index - 1; i < index + M; i += M) {
                for (j = 1; j < 3; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
        case 6:         //top right
            for (i = index - 1; i < index + M; i += M) {
                for (j = 0; j < 2; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
        case 7:         //bottom left
            for (i = index - M - 1; i < index; i += M) {
                for (j = 1; j < 3; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
        default:        //bottom right
            for (i = index - M - 1; i < index; i += M) {
                for (j = 0; j < 2; j++) {
                    if (*(system + i + j)) {
                        neighbors++;
                    }
                }
            }
            break;
    }
    
    if (*(system + index) && neighbors <= 1) {
        temp = 0;
    } else if (*(system + index) && neighbors >= 4) {
        temp = 0;
    } else if (!*(system + index) && (neighbors == 2 || neighbors == 3)) {
        temp = 1;
    } else {
        temp = *(system + index);
    }
    
    __syncthreads();
    *(system + index) = temp;
}

void genArray(bool *array, int M, int N) {
    int i, j;
    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            if (rand()%2 == 0) {
                *(array + i*M + j) = 1;
            } else {
                *(array + i*M + j) = 0;
            }
        }
    }
}

void printArray(bool *array, int M, int N) {
    int i, j;
    
    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            printf("%d ", *(array + i*M + j));
        }
        printf("\n");
    }
}

int main() {
    int i, j, M, N, K, iteration, size, execTime = 0, population;
    bool *system, debug = 0, *d_system;
    std::chrono::time_point<std::chrono::high_resolution_clock> gpuStart, gpuEnd;
    
    printf("Enter the number of rows (M): ");
    if (scanf("%d", &M)) {}
    printf("Enter the number of columns (N): ");
    if (scanf("%d", &N)) {}
    printf("Enter the number of iterations (K): ");
    if (scanf("%d", &K)) {}

    size = M * N;
    srand(time(NULL));

    system = (bool*)malloc(size * sizeof(bool*));
    
    hipMalloc((void**)&d_system, size * sizeof(bool));


    genArray(system, M, N);
    printf("Initial System State:\n");
    printArray(system, M, N);

    population = 0;
    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            population += (int)*(system + i * M + j);
        }
    }
    printf("Population Density: %0.2f\n", 1.0 * population / size);

    hipMemcpy(d_system, system, size*sizeof(bool), hipMemcpyHostToDevice);

    
    for (iteration = 1; iteration <= K; iteration++) {
        population = 0;
        gpuStart = std::chrono::high_resolution_clock::now();
        manipulateSystem <<<M, N>>> (d_system, M, N);
        gpuEnd = std::chrono::high_resolution_clock::now();
        auto gpuTime = std::chrono::duration_cast<std::chrono::microseconds>(gpuEnd - gpuStart);
        execTime += gpuTime.count();
        hipMemcpy(system, d_system, size*sizeof(bool), hipMemcpyDeviceToHost);

        if (debug) {
            printf("iteration: %d\n", iteration);
            printArray(system, M, N);
            printf("Execution time: %d microseconds\n", gpuTime.count());
            for (i = 0; i < M; i++) {
                for (j = 0; j < N; j++) {
                    population += (int) *(system + i * M + j);
                }
            }
            printf("Population Density: %0.2f\n", 1.0 * population / size);
        }
    }

    printf("\nResult:\n");
    printArray(system, M, N);
    printf("Total Execution Time: %d microseconds\n", execTime);
    population = 0;
    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            population += (int)*(system + i * M + j);
        }
    }
    printf("Population Density: %0.2f\n", 1.0 * population / size);

    free(system);
    hipFree(d_system);
    return 0;
}
